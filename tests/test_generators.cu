#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include "util.h"
#include "util.hpp"
#include "util.cuh"

#include "cpgplot.h"

#include <catch2/catch_test_macros.hpp>

#define STRLEN 256

using namespace std;

int find_maxmin(float *data, float &datamin, float &datamax, int ndata){

  datamin = data[0];
  datamax = data[0];
  
  for(int i = 1; i < ndata; i++){
    datamin = (datamin>data[i]) ? data[i] : datamin;
    datamax = (datamax<data[i]) ? data[i] : datamax;
  }

  return EXIT_SUCCESS;
}

int create_x(float xmin, float xmax, int ndata, float *x){
  
  for(int i = 0; i < ndata; i++){
    x[i] = xmin + i*(xmax-xmin)/(float)ndata;
  }
  
  return EXIT_SUCCESS;
}

int create_y(unsigned *y_int, float *y_float, int ndata){
  
  for(int i = 0; i < ndata; i++){
    y_float[i] = y_int[i];
  }
  
  return EXIT_SUCCESS;
}

int plotit(unsigned *data, float min, float max, char *device, char *title){

  float x[NUM_BINS];
  float y[NUM_BINS];
  create_x(min, max, NUM_BINS, x);
  create_y(data, y, NUM_BINS);

  float ymax;
  float ymin;
  find_maxmin(y, ymin, ymax, NUM_BINS);
  
  /* Open graphics device. */
  if (cpgopen(device) < 1){
    //if (cpgopen("/xw") < 1){
    fprintf(stderr, "Can not open device to plot\n");
    exit(1);
  }

  /* Get rid of  Press RETURN for next page:  */
  cpgask(0);

  /* Axis ranges */
  cpgenv(x[0], x[NUM_BINS-1], ymin, ymax, 0, 0);

  /* Label the axes (note use of \\u and \\d for raising exponent). */
  cpglab("Sample Value", "Number of Samples", title);

  /* plot histogram */
  cpgpt(NUM_BINS, x, y, 1);

  /* Close plot figure */
  cpgclos();
  
  return EXIT_SUCCESS;
}

TEST_CASE("RealDataGeneratorUniform", "RealDataGeneratorUniform") {

  int nthread = 128;
  int ndata = 102400000;
  int exclude = 0;
  int include = 100;

  hipEvent_t g_start;
  hipEvent_t g_stop;
  float gtime = 0;
  checkCudaErrors(hipEventCreate(&g_start));
  checkCudaErrors(hipEventCreate(&g_stop));
  CUDA_STARTTIME(g);
  
  // Get data
  hiprandGenerator_t gen;
  checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));
  RealDataGeneratorUniform uniform_data(gen, ndata, exclude, include, nthread);
  print_cuda_memory_info();

  // Get mean and standard deviation
  RealDataMeanStddevCalcultor<float> mean_stddev(uniform_data.data, ndata, nthread, 7);
  cout << "uniform data mean is " << mean_stddev.mean << "\t"
       << "uniform data stddev is " << mean_stddev.stddev 
       << endl;

  // Get histogram
  float min = exclude;
  float max = include;
  int nblock = 256;
  RealDataHistogram<float> histogram(uniform_data.data, ndata, min, max, nblock, nthread);

  CUDA_STOPTIME(g);
  cout << "elapsed time is " << gtime << " milliseconds" << endl;
  
  // plot histogram
  char device[STRLEN];
  char title[STRLEN];
  strcpy(device, "uniform.ps/ps");
  strcpy(title,  "Uniform Distribution");
  plotit(histogram.data, min, max, device, title);
}    

TEST_CASE("RealDataGeneratorNormal", "RealDataGeneratorNormal") {

  int ndata = 102400000;
  float mean = 0;
  float stddev = 10;
  int nthread = 128;

  hipEvent_t g_start;
  hipEvent_t g_stop;
  float gtime = 0;
  checkCudaErrors(hipEventCreate(&g_start));
  checkCudaErrors(hipEventCreate(&g_stop));
  CUDA_STARTTIME(g);
  
  // Get data
  hiprandGenerator_t gen;
  checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));
  RealDataGeneratorNormal normal_data(gen, mean, stddev, ndata);
  print_cuda_memory_info();

  // Get mean and standard deviation
  RealDataMeanStddevCalcultor<float> mean_stddev(normal_data.data, ndata, nthread, 7);
  cout << "normal data mean is " << mean_stddev.mean << "\t"
       << "normal data stddev is " << mean_stddev.stddev 
       << endl;

  // Get histogram
  float min = -50;
  float max = 50;
  int nblock = 256;
  RealDataHistogram<float> histogram(normal_data.data, ndata, min, max, nblock, nthread);

  CUDA_STOPTIME(g);
  cout << "elapsed time is " << gtime << " milliseconds" << endl;
  
  // plot histogram
  char device[STRLEN];
  char title[STRLEN];
  strcpy(device, "normal.ps/ps");
  strcpy(title,  "Normal Distribution");
  plotit(histogram.data, min, max, device, title);
}
