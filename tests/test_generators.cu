#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include "util.h"
#include "util.hpp"
#include "util.cuh"

#include <catch2/catch_test_macros.hpp>

using namespace std;

TEST_CASE("RealDataGeneratorUniform", "RealDataGeneratorUniform") {

  int nthread = 128;
  int ndata = 10240;
  int exclude = 10;
  int include = 100;
  
  hiprandGenerator_t gen;
  checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

  RealDataGeneratorUniform uniform_data(gen, ndata, exclude, include, nthread);

  for(int i = 0; i < ndata; i++){
    cout << uniform_data.data[i] << endl;
  }

  // I also need to add histogram here for a better check
}    

TEST_CASE("RealDataGeneratorNormal", "RealDataGeneratorNormal") {

  int ndata = 10240;
  float mean = 10;
  float stddev = 10;
  
  hiprandGenerator_t gen;
  checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));

  RealDataGeneratorNormal normal_data(gen, mean, stddev, ndata);

  for(int i = 0; i < ndata; i++){
    cout << normal_data.data[i] << endl;
  }

  // I also need to add histogram here for a better check
}    
