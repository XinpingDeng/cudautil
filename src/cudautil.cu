#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include "../include/cudautil.h"
#include "../include/reduction_kernel.h"

int print_cuda_memory_info() {
  //hipError_t status;
  size_t free, total;
  
  checkCudaErrors(hipMemGetInfo(&free, &total));
  
  fprintf(stdout, "GPU free memory is %.1f, total is %.1f MBbytes\n",
	  free/1024.0/1024, total/1024.0/1024);
  
  if(free<=0){
    fprintf(stderr, "Use too much GPU memory.\n");
    exit(EXIT_FAILURE);
  }
  
  return EXIT_SUCCESS;
}

/*! A kernel to contraint random number from range (0.0 1.0] to range (exclude include] or [include exclude).
 *
 * \param[in, out] data    The input data in range (0.0 1.0] and new data in range (exclude include] or [include exclude) is also returned with it.
 * \param[in]      exclude The exclusive end of random numbers
 * \param[in]      range   The range of random numbers, it does not have to be positive, it is calculated with `include - exclude`
 * \param[in]      ndata   Number of data
 *
 */
__global__ void cudautil_contraintor(float *data, float exclude, float range, int ndata){
  // Maximum x-dimension of a grid of thread blocks is 2^31-1
  // Maximum x- or y-dimension of a block is 1024
  // So here we can cover (2^31-1)*1024 random numbers, which are 2^41-1024
  // should be big enough

  int idx = blockDim.x*blockIdx.x + threadIdx.x;
  if(idx<ndata){
    // Just in case we have a very small ndata
    data[idx] = data[idx]*range+exclude;
  }
}
