#ifndef _GNU_SOURCE
#define _GNU_SOURCE
#endif

#include "utils/cuda_utils.h"

using namespace std;

// We can not compare converted numbers against CPU implementation
// We can only check the difference between original data and converted data with mean and standard deviation
int main(int argc, char *argv[]){

  int ndata = 102400000;
  float mean = 0;
  float stddev = 10;
  int nthread = 128;

  hipEvent_t g_start;
  hipEvent_t g_stop;
  float gtime = 0;
  checkCudaErrors(hipEventCreate(&g_start));
  checkCudaErrors(hipEventCreate(&g_stop));
  CUDA_STARTTIME(g);
  
  // Get float data
  hiprandGenerator_t gen;
  checkCudaErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));
  RealGeneratorNormal normal_data(gen, mean, stddev, ndata);
  print_cuda_memory_info();

  // Convert to float
  RealConvertor<float, float> normal_data_float(normal_data.data, ndata, nthread);

  // Get the difference
  RealDifferentiator<float, float> normal_data_diff(normal_data.data, normal_data_float.data, ndata, nthread);
  
  // Get mean and standard deviation
  RealMeanStddevCalculator<float> mean_stddev(normal_data_diff.data, ndata, nthread, 7);
  
  CUDA_STOPTIME(g);
  cout << "elapsed time is " << gtime << " milliseconds" << endl;

  float mean_f = mean_stddev.mean;
  float stddev_f = mean_stddev.stddev ;
  cout << "mean is " << mean_f << " "
       << "stddev is " << stddev_f 
       << endl;

  return EXIT_SUCCESS;
}
